#include "hip/hip_runtime.h"
#include "cuda_func.h"

/*
 * Функция ядра.
 * Расчет теплоемкости на видеокарте.
 * Логика функции аналогична функции для центрального процессора.
 */
/*__global__ void calc ( double* matrix, double* newMatrix, int length, int width, double dX2, double dY2, double dT, int **recieveBorderType, double** externalBorder, int* externalBorderMove ) {
	double top, left, bottom, right, cur;

	int i = BLOCK_LENGHT_SIZE * blockIdx.x + threadIdx.x;
	int j = BLOCK_WIDTH_SIZE * blockIdx.y + threadIdx.y;

	if( i < length && j < width ) {
		if( i == 0 )
			if( recieveBorderType[TOP][j] == BY_FUNCTION ) {
				newMatrix[i * width + j] = 100;
				return;
			}
			else
				top = externalBorder[	recieveBorderType[TOP][j]	][j - externalBorderMove[	recieveBorderType[TOP][j]	]];
		else
			top = matrix[(i - 1) * width + j];
	
	
		if( j == 0 )
			if( recieveBorderType[LEFT][i] == BY_FUNCTION ) {
				newMatrix[i * width + j] = 10;
				return;
			}
			else
				left = externalBorder[	recieveBorderType[LEFT][i]	][i - externalBorderMove[	recieveBorderType[LEFT][i]		]];
		else
			left = matrix[i * width + (j - 1)];
	
	
		if( i == length - 1 )
			if( recieveBorderType[BOTTOM][j] == BY_FUNCTION ) {
				newMatrix[i * width + j] = 10;
				return;
			}
			else
				bottom = externalBorder[	recieveBorderType[BOTTOM][j]	][j - externalBorderMove[	recieveBorderType[BOTTOM][j]	]];
		else
			bottom = matrix[(i + 1) * width + j];
	
	
		if( j == width - 1 )
			if( recieveBorderType[RIGHT][i] == BY_FUNCTION ) {
				newMatrix[i * width + j] = 10;
				return;
			}
			else
				right = externalBorder[	recieveBorderType[RIGHT][i]	][i - externalBorderMove[	recieveBorderType[RIGHT][i]	]];
		else
			right = matrix[i * width + (j + 1)];

	
		cur = matrix[i * width + j];
	
		newMatrix[i * width + j] = cur + dT * ( ( left - 2*cur + right )/dX2 + ( top - 2*cur + bottom )/dY2  );
	}
}*/

/*__global__ void calcBorder ( double* matrix, double* newMatrix, int length, int width, double dX2, double dY2, double dT, int **recieveBorderType, double** externalBorder, int* externalBorderMove ) {
	double top, left, bottom, right, cur;

	int i = BLOCK_LENGHT_SIZE * blockIdx.x + threadIdx.x;
	int j = BLOCK_WIDTH_SIZE * blockIdx.y + threadIdx.y;

	if( i < length && j < width )
		if( i == 0 || i == length - 1 || j == 0 || j == width - 1 ) {
			if( i == 0 )
				if( recieveBorderType[TOP][j] == BY_FUNCTION ) {
					newMatrix[i * width + j] = 100;
					return;
				}
				else
					top = externalBorder[	recieveBorderType[TOP][j]	][j - externalBorderMove[	recieveBorderType[TOP][j]	]];
			else
				top = matrix[(i - 1) * width + j];
		
		
			if( j == 0 )
				if( recieveBorderType[LEFT][i] == BY_FUNCTION ) {
					newMatrix[i * width + j] = 10;
					return;
				}
				else
					left = externalBorder[	recieveBorderType[LEFT][i]	][i - externalBorderMove[	recieveBorderType[LEFT][i]		]];
			else
				left = matrix[i * width + (j - 1)];
		
		
			if( i == length - 1 )
				if( recieveBorderType[BOTTOM][j] == BY_FUNCTION ) {
					newMatrix[i * width + j] = 10;
					return;
				}
				else
					bottom = externalBorder[	recieveBorderType[BOTTOM][j]	][j - externalBorderMove[	recieveBorderType[BOTTOM][j]	]];
			else
				bottom = matrix[(i + 1) * width + j];
		
		
			if( j == width - 1 )
				if( recieveBorderType[RIGHT][i] == BY_FUNCTION ) {
					newMatrix[i * width + j] = 10;
					return;
				}
				else
					right = externalBorder[	recieveBorderType[RIGHT][i]	][i - externalBorderMove[	recieveBorderType[RIGHT][i]	]];
			else
				right = matrix[i * width + (j + 1)];
		
		
			cur = matrix[i * width + j];
		
			newMatrix[i * width + j] = cur + dT * ( ( left - 2*cur + right )/dX2 + ( top - 2*cur + bottom )/dY2  );
		}
}*/

/*__global__ void calcCenter ( double* matrix, double* newMatrix, int length, int width, double dX2, double dY2, double dT, int **recieveBorderType, double** externalBorder, int* externalBorderMove ) {
	double top, left, bottom, right, cur;

	int i = BLOCK_LENGHT_SIZE * blockIdx.x + threadIdx.x;
	int j = BLOCK_WIDTH_SIZE * blockIdx.y + threadIdx.y;
	
	if( (i > 1) && (i < length - 1) && (j > 1) && (j < width - 1) ) {
		top = matrix[(i - 1) * width + j];
		left = matrix[i * width + (j - 1)];
		bottom = matrix[(i + 1) * width + j];
		right = matrix[i * width + (j + 1)];

		cur = matrix[i * width + j];
	
		newMatrix[i * width + j] = cur + dT * ( ( left - 2*cur + right )/dX2 + ( top - 2*cur + bottom )/dY2  );
	}
	
	if( i < length && j < width )
		if( i != 0 && i != length - 1 && j != 0 && j != width - 1 ) {
			top = matrix[(i - 1) * width + j];
			left = matrix[i * width + (j - 1)];
			bottom = matrix[(i + 1) * width + j];
			right = matrix[i * width + (j + 1)];
	
			cur = matrix[i * width + j];
		
			newMatrix[i * width + j] = cur + dT * ( ( left - 2*cur + right )/dX2 + ( top - 2*cur + bottom )/dY2  );
		}
}*/

__global__ void assignIntArray (int* array, int value, int arrayLength) {
	int	idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	
	if( idx < arrayLength )
		array[idx] = value;
}

__global__ void copyIntArray (int* dest, int* source, int arrayLength) {
	int	idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	
	if( idx < arrayLength )
		dest[idx] = source[idx];
}

__global__ void assignDoubleArray (double* array, double value, int arrayLength) {
	int	idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	
	if( idx < arrayLength )
		array[idx] = value;
}

__global__ void copyDoubleArray (double* dest, double* source, int arrayLength) {
	int	idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	
	if( idx < arrayLength )
		dest[idx] = source[idx];
}

__global__ void sumDoubleArray (double* arg1, double* arg2, double* result, int arrayLength) {
	int	idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	
	if( idx < arrayLength )
		result[idx] = arg1[idx] + arg2[idx];
}

__global__ void multiplyDoubleArrayByNumber (double* array, double value, double* result, int arrayLength) {
	int	idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	
	if( idx < arrayLength )
		result[idx] = array[idx] * value;
}



__global__ void multiplyByNumberAndSumDoubleArrays(double* array1, double value1, double* array2, double value2, double* result, int arrayLength) {
	int	idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	
	if( idx < arrayLength )
		result[idx] = value1 * array1[idx] + value2 * array2[idx];
}

__global__ void multiplyByNumberAndSumDoubleArrays(double* array1, double value1, double* array2, double value2, double* array3, double value3, double* result, int arrayLength) {
	int	idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	
	if( idx < arrayLength )
		result[idx] = value1 * array1[idx] + value2 * array2[idx] + value3 * array3[idx];
}

__global__ void multiplyByNumberAndSumDoubleArrays(double* array1, double value1, double* array2, double value2, double* array3, double value3, double* array4, double value4, double* result, int arrayLength) {
	int	idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	
	if( idx < arrayLength )
		result[idx] = value1 * array1[idx] + value2 * array2[idx] + value3 * array3[idx] + value4 * array4[idx];
}

__global__ void multiplyByNumberAndSumDoubleArrays(double* array1, double value1, double* array2, double value2, double* array3, double value3, double* array4, double value4, double* array5, double value5, double* result, int arrayLength) {
	int	idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	
	if( idx < arrayLength )
		result[idx] = value1 * array1[idx] + value2 * array2[idx] + value3 * array3[idx] + value4 * array4[idx] + value5 * array5[idx];
}

__global__ void multiplyByNumberAndSumDoubleArrays(double* array1, double value1, double* array2, double value2, double* array3, double value3, double* array4, double value4, double* array5, double value5, double* array6, double value6, double* result, int arrayLength) {
	int	idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	
	if( idx < arrayLength )
		result[idx] = value1 * array1[idx] + value2 * array2[idx] + value3 * array3[idx] + value4 * array4[idx] + value5 * array5[idx] + value6 * array6[idx];
}


__global__ void sumElementOfDoubleArray(double* array, double* result, int arrayLength) {
    __shared__ double data[BLOCK_SIZE];
    
    int tid=threadIdx.x; 
    int idx=blockIdx.x*blockDim.x+threadIdx.x;
    
    data[tid] = ( idx < arrayLength ) ? array[idx] : 0;

    __syncthreads();// ждем пока все нити(потоки) скопируют данные. 
 
    for(int s = blockDim.x/2; s > 0; s = s/2) { 
        if (tid < s)
        	data[tid] += data[ tid + s ]; 
        __syncthreads(); 
    }
    
    if ( tid==0 ) 
        result[blockIdx.x] = data[0]; 
}

__global__ void forGetStepErrorDP45(double* mTempStore1, double e1,
		double* mTempStore3, double e3, double* mTempStore4, double e4,
		double* mTempStore5, double e5, double* mTempStore6, double e6,
		double* mTempStore7, double e7, double* mState, double* mArg,
		double timeStep, double aTol, double rTol, double mCount, double* result) {
	int	idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
		
	if( idx < mCount ) {
		
		mTempStore1[idx] = timeStep * (e1 * mTempStore1[idx] + e3 * mTempStore3[idx] + e4 * mTempStore4[idx] + e5 * mTempStore5[idx] + e6 * mTempStore6[idx]+ e7 * mTempStore7[idx]) /
				(aTol + rTol * max(mArg[idx], mState[idx]));
				
		result[idx] = mTempStore1[idx] * mTempStore1[idx];		
	}
}



__global__ void prepareBorderDevice(double* dest, double* source, int borderNumber, int zCount, int yCount, int xCount) {
	
}


void assignArray(int* array, int value, int arrayLength) {
	dim3 threads ( BLOCK_SIZE );
	dim3 blocks  ( (int)ceil((double)arrayLength / threads.x) );
	
	assignIntArray <<< blocks, threads >>> ( array, value, arrayLength);
}

void assignArray(double* array, double value, int arrayLength) {
	dim3 threads ( BLOCK_SIZE );
	dim3 blocks  ( (int)ceil((double)arrayLength / threads.x) );
	
	assignDoubleArray <<< blocks, threads >>> ( array, value, arrayLength);
}

void copyArray(int* dest, int* source, int arrayLength) {
	dim3 threads ( BLOCK_SIZE );
	dim3 blocks  ( (int)ceil((double)arrayLength / threads.x) );
	
	copyIntArray <<< blocks, threads >>> ( dest, source, arrayLength);
}


void copyArray(double* dest, double* source, int arrayLength) {
	dim3 threads ( BLOCK_SIZE );
	dim3 blocks  ( (int)ceil((double)arrayLength / threads.x) );
	
	copyDoubleArray <<< blocks, threads >>> ( dest, source, arrayLength);
}

void sumArray(double* arg1, double* arg2, double* result, int arrayLength) {
	dim3 threads ( BLOCK_SIZE );
	dim3 blocks  ( (int)ceil((double)arrayLength / threads.x) );
	
	sumDoubleArray <<< blocks, threads >>> ( arg1, arg2, result, arrayLength);
}

void multiplyArrayByNumber(double* array, double value, double* result, int arrayLength) {
	dim3 threads ( BLOCK_SIZE );
	dim3 blocks  ( (int)ceil((double)arrayLength / threads.x) );
	
	multiplyDoubleArrayByNumber <<< blocks, threads >>> ( array, value, result, arrayLength);
}


void multiplyByNumberAndSumArrays(double* array1, double value1, double* array2, double value2, double* result, int arrayLength) {
	dim3 threads ( BLOCK_SIZE );
	dim3 blocks  ( (int)ceil((double)arrayLength / threads.x) );
	
	multiplyByNumberAndSumDoubleArrays <<< blocks, threads >>> ( array1, value1, array2, value2, result, arrayLength);
}

void multiplyByNumberAndSumArrays(double* array1, double value1, double* array2, double value2, double* array3, double value3, double* result, int arrayLength) {
	dim3 threads ( BLOCK_SIZE );
	dim3 blocks  ( (int)ceil((double)arrayLength / threads.x) );
	
	multiplyByNumberAndSumDoubleArrays <<< blocks, threads >>> ( array1, value1, array2, value2, array3, value3, result, arrayLength);
}

void multiplyByNumberAndSumArrays(double* array1, double value1, double* array2, double value2, double* array3, double value3, double* array4, double value4, double* result, int arrayLength) {
	dim3 threads ( BLOCK_SIZE );
	dim3 blocks  ( (int)ceil((double)arrayLength / threads.x) );
	
	multiplyByNumberAndSumDoubleArrays <<< blocks, threads >>> ( array1, value1, array2, value2, array3, value3, array4, value4, result, arrayLength);
}

void multiplyByNumberAndSumArrays(double* array1, double value1, double* array2, double value2, double* array3, double value3, double* array4, double value4, double* array5, double value5, double* result, int arrayLength) {
	dim3 threads ( BLOCK_SIZE );
	dim3 blocks  ( (int)ceil((double)arrayLength / threads.x) );
	
	multiplyByNumberAndSumDoubleArrays <<< blocks, threads >>> ( array1, value1, array2, value2, array3, value3, array4, value4, array5, value5, result, arrayLength);
}

void multiplyByNumberAndSumArrays(double* array1, double value1, double* array2, double value2, double* array3, double value3, double* array4, double value4, double* array5, double value5, double* array6, double value6, double* result, int arrayLength) {
	dim3 threads ( BLOCK_SIZE );
	dim3 blocks  ( (int)ceil((double)arrayLength / threads.x) );
	
	multiplyByNumberAndSumDoubleArrays <<< blocks, threads >>> ( array1, value1, array2, value2, array3, value3, array4, value4, array5, value5, array6, value6, result, arrayLength);
}




void prepareBorder(double* dest, double* source, int borderNumber, int zCount, int yCount, int xCount) {
	printf("\nPreapre border GPU\n");
}

void computeCenter() {
	printf("\nCompute center GPU\n");
}

void computeBorder() {
	printf("\nCompute border GPU\n");
}

double sumElementOfArray(double* array, int arrayLength) {
	double sumHost;
	double* sumDevice;
	
	hipMalloc( (void**)&sumDevice, 1 * sizeof(double) );
	
	dim3 threads ( BLOCK_SIZE );
	dim3 blocks  ( (int)ceil((double)arrayLength / threads.x) );
		
	sumElementOfDoubleArray <<< blocks, threads >>> ( array, sumDevice, arrayLength );
	
	hipMemcpy(&sumHost, sumDevice, 1 * sizeof(double), hipMemcpyDeviceToHost);
	hipFree(sumDevice);
	
	return sumHost;
}

double getStepErrorDP45(double* mTempStore1, double e1,
		double* mTempStore3, double e3, double* mTempStore4, double e4,
		double* mTempStore5, double e5, double* mTempStore6, double e6,
		double* mTempStore7, double e7, double* mState, double* mArg,
		double timeStep, double aTol, double rTol, double mCount) {
	
	double errorHost;
	double* errorDevice;
	
	hipMalloc( (void**)&errorDevice, 1 * sizeof(double) );
	
	dim3 threads ( BLOCK_SIZE );
	dim3 blocks  ( (int)ceil((double)mCount / threads.x) );
		
	forGetStepErrorDP45 <<< blocks, threads >>> ( mTempStore1, e1, mTempStore3, e3, mTempStore4, e4, mTempStore5, e5, mTempStore6, e6, mTempStore7, e7, mState, mArg, timeStep, aTol, rTol, mCount, mTempStore1 );
	sumElementOfDoubleArray <<< blocks, threads >>> ( mTempStore1, errorDevice, mCount );
	
	hipMemcpy(&errorHost, errorDevice, 1 * sizeof(double), hipMemcpyDeviceToHost);
	hipFree(errorDevice);
	
	return errorHost;
}
/*
 * Функция ядра
 * Копирование данных из матрицы в массив.
 * Используется при подготовке пересылаемых данных.
 */
/*__global__ void copyBorderFromMatrix ( double** blockBorder, double* matrix, int** sendBorderType, int* blockBorderMove, int side, int length, int width ) {
	int idx  = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	
	if( (side == TOP || side == BOTTOM) && idx >= width )
		return;
	
	if( (side == LEFT || side == RIGHT) && idx >= length )
		return;

	if( sendBorderType[side][idx] == BY_FUNCTION )
		return;
	
	double value;
	
	switch (side) {
		case TOP:
			value = matrix[0 * width + idx];
			break;
		case LEFT:
			value = matrix[idx * width + 0];
			break;
		case BOTTOM:
			value = matrix[(length - 1) * width + idx];
			break;
		case RIGHT:
			value = matrix[idx * width + (width - 1)];
			break;
		default:
			break;
	}
	
	blockBorder[	sendBorderType[side][idx]	][idx - blockBorderMove[	sendBorderType[side][idx]	]] = value;
}*/