#include "hip/hip_runtime.h"
/*
 * BlockGpu.cpp
 *
 *  Created on: 29 янв. 2015 г.
 *      Author: frolov
 */

#include "blockgpu.h"

/*
 * Функция ядра.
 * Расчет теплоемкости на видеокарте.
 * Логика функции аналогична функции для центрального процессора.
 */
__global__ void calc ( double* matrix, double* newMatrix, int length, int width, double dX2, double dY2, double dT, int **recieveBorderType, double** externalBorder, int* externalBorderMove ) {

	double top, left, bottom, right, cur;

	int i = BLOCK_LENGHT_SIZE * blockIdx.x + threadIdx.x;
	int j = BLOCK_WIDTH_SIZE * blockIdx.y + threadIdx.y;

	if( i < length && j < width ) {
		if( i == 0 )
			if( recieveBorderType[TOP][j] == BY_FUNCTION ) {
				newMatrix[i * width + j] = 100;
				return;
			}
			else
				top = externalBorder[	recieveBorderType[TOP][j]	][j - externalBorderMove[	recieveBorderType[TOP][j]	]];
		else
			top = matrix[(i - 1) * width + j];
	
	
		if( j == 0 )
			if( recieveBorderType[LEFT][i] == BY_FUNCTION ) {
				newMatrix[i * width + j] = 10;
				return;
			}
			else
				left = externalBorder[	recieveBorderType[LEFT][i]	][i - externalBorderMove[	recieveBorderType[LEFT][i]		]];
		else
			left = matrix[i * width + (j - 1)];
	
	
		if( i == length - 1 )
			if( recieveBorderType[BOTTOM][j] == BY_FUNCTION ) {
				newMatrix[i * width + j] = 10;
				return;
			}
			else
				bottom = externalBorder[	recieveBorderType[BOTTOM][j]	][j - externalBorderMove[	recieveBorderType[BOTTOM][j]	]];
		else
			bottom = matrix[(i + 1) * width + j];
	
	
		if( j == width - 1 )
			if( recieveBorderType[RIGHT][i] == BY_FUNCTION ) {
				newMatrix[i * width + j] = 10;
				return;
			}
			else
				right = externalBorder[	recieveBorderType[RIGHT][i]	][i - externalBorderMove[	recieveBorderType[RIGHT][i]	]];
		else
			right = matrix[i * width + (j + 1)];
	
	
		cur = matrix[i * width + j];
	
		newMatrix[i * width + j] = cur + dT * ( ( left - 2*cur + right )/dX2 + ( top - 2*cur + bottom )/dY2  );
	}
}

/*
 * Функция ядра
 * Заполнение целочисленного массива определенным значением.
 */
__global__ void assignIntArray (int* arr, int value, int arrayLength) {
	int	idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	
	if( idx < arrayLength )
		arr[idx] = value;
}

/*
 * Функция ядра
 * Копирование целочесленных массивов.
 */
__global__ void copyIntArray (int* dest, int* source, int arrayLength) {
	int	idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	
	if( idx < arrayLength )
		dest[idx] = source[idx];
}

/*
 * Функция ядра
 * Заполнение вещественного массива определенным значением.
 */
__global__ void assignDoubleArray (double* arr, double value, int arrayLength) {
	int	idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	
	if( idx < arrayLength )
		arr[idx] = value;
}

/*
 * Функция ядра
 * Копирование данных из матрицы в массив.
 * Используется при подготовке пересылаемых данных.
 */
__global__ void copyBorderFromMatrix ( double** blockBorder, double* matrix, int** sendBorderType, int* blockBorderMove, int side, int length, int width )
{
	int idx  = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	
	if( (side == TOP || side == BOTTOM) && idx >= width )
		return;
	
	if( (side == LEFT || side == RIGHT) && idx >= length )
		return;

	if( sendBorderType[side][idx] == BY_FUNCTION )
		return;
	
	double value;
	
	switch (side) {
		case TOP:
			value = matrix[0 * width + idx];
			break;
		case LEFT:
			value = matrix[idx * width + 0];
			break;
		case BOTTOM:
			value = matrix[(length - 1) * width + idx];
			break;
		case RIGHT:
			value = matrix[idx * width + (width - 1)];
			break;
		default:
			break;
	}
	
	blockBorder[	sendBorderType[side][idx]	][idx - blockBorderMove[	sendBorderType[side][idx]	]] = value;
}

BlockGpu::BlockGpu(int _length, int _width, int _lengthMove, int _widthMove, int _nodeNumber, int _deviceNumber) : Block(  _length, _width, _lengthMove, _widthMove, _nodeNumber, _deviceNumber ) {
	deviceNumber = _deviceNumber;
	
	hipSetDevice(deviceNumber);
	
	dim3 threads ( BLOCK_SIZE );
	dim3 blocksLength  ( (int)ceil((double)length / threads.x) );
	dim3 blocksWidth  ( (int)ceil((double)width / threads.x) );
	dim3 blocksLengthWidth ( (int)ceil((double)(length * width) / threads.x) );
	
	hipMalloc( (void**)&matrix, width * length * sizeof(double) );
	hipMalloc( (void**)&newMatrix, width * length * sizeof(double) );
	
	assignDoubleArray <<< blocksLengthWidth, threads >>> ( matrix, 0, length * width);
	assignDoubleArray <<< blocksLengthWidth, threads >>> ( newMatrix, 0, length * width);

	/*
	 * Типы границ блока. Выделение памяти.
	 */
	sendBorderType = new int* [BORDER_COUNT];

	hipMalloc ( (void**)&sendBorderType[TOP], width * sizeof(int) );
	assignIntArray <<< blocksWidth, threads >>> ( sendBorderType[TOP], BY_FUNCTION, width ); 

	hipMalloc ( (void**)&sendBorderType[LEFT], length * sizeof(int) );
	assignIntArray <<< blocksLength, threads >>> ( sendBorderType[LEFT], BY_FUNCTION, length );

	hipMalloc ( (void**)&sendBorderType[BOTTOM], width * sizeof(int) );
	assignIntArray <<< blocksWidth, threads >>> ( sendBorderType[BOTTOM], BY_FUNCTION, width ); 

	hipMalloc ( (void**)&sendBorderType[RIGHT], length * sizeof(int) );
	assignIntArray <<< blocksLength, threads >>> ( sendBorderType[RIGHT], BY_FUNCTION, length );
	
	hipMalloc ( (void**)&sendBorderTypeOnDevice, BORDER_COUNT * sizeof(int*) );
	hipMemcpy( sendBorderTypeOnDevice, sendBorderType, BORDER_COUNT * sizeof(int*), hipMemcpyHostToDevice );
	
	
	receiveBorderType = new int* [BORDER_COUNT];

	hipMalloc ( (void**)&receiveBorderType[TOP], width * sizeof(int) );
	assignIntArray <<< blocksWidth, threads >>> ( receiveBorderType[TOP], BY_FUNCTION, width ); 

	hipMalloc ( (void**)&receiveBorderType[LEFT], length * sizeof(int) );
	assignIntArray <<< blocksLength, threads >>> ( receiveBorderType[LEFT], BY_FUNCTION, length );

	hipMalloc ( (void**)&receiveBorderType[BOTTOM], width * sizeof(int) );
	assignIntArray <<< blocksWidth, threads >>> ( receiveBorderType[BOTTOM], BY_FUNCTION, width ); 

	hipMalloc ( (void**)&receiveBorderType[RIGHT], length * sizeof(int) );
	assignIntArray <<< blocksLength, threads >>> ( receiveBorderType[RIGHT], BY_FUNCTION, length );
	
	hipMalloc ( (void**)&receiveBorderTypeOnDevice, BORDER_COUNT * sizeof(int*) );
	hipMemcpy( receiveBorderTypeOnDevice, receiveBorderType, BORDER_COUNT * sizeof(int*), hipMemcpyHostToDevice );
	
	result = new double [length * width];
}

BlockGpu::~BlockGpu() {
	if(matrix != NULL)
		hipFree(matrix);
	
	if(newMatrix != NULL)
		hipFree(newMatrix);
	
	if(sendBorderType != NULL) {
		if(sendBorderType[TOP] != NULL)
			hipFree(sendBorderType[TOP]);
		
		if(sendBorderType[LEFT] != NULL)
			hipFree(sendBorderType[LEFT]);
		
		if(sendBorderType[BOTTOM] != NULL)
			hipFree(sendBorderType[BOTTOM]);
		
		if(sendBorderType[RIGHT] != NULL)
			hipFree(sendBorderType[RIGHT]);
		
		hipFree(sendBorderTypeOnDevice);
		delete sendBorderType;
	}
	
	if(receiveBorderType != NULL) {
		if(receiveBorderType[TOP] != NULL)
			hipFree(receiveBorderType[TOP]);
		
		if(receiveBorderType[LEFT] != NULL)
			hipFree(receiveBorderType[LEFT]);
		
		if(receiveBorderType[BOTTOM] != NULL)
			hipFree(receiveBorderType[BOTTOM]);
		
		if(receiveBorderType[RIGHT] != NULL)
			hipFree(receiveBorderType[RIGHT]);
		
		hipFree(receiveBorderTypeOnDevice);
		delete receiveBorderType;
	}
	
	if(result != NULL)
		delete result;
}

void BlockGpu::computeOneStep(double dX2, double dY2, double dT) {
	hipSetDevice(deviceNumber);
	
	dim3 threads ( BLOCK_LENGHT_SIZE, BLOCK_WIDTH_SIZE );
	dim3 blocks  ( (int)ceil((double)length / threads.x), (int)ceil((double)width / threads.y) );

	calc <<< blocks, threads >>> ( matrix, newMatrix, length, width, dX2, dY2, dT, receiveBorderTypeOnDevice, externalBorderOnDevice, externalBorderMove );
	
	double* tmp = matrix;

	matrix = newMatrix;

	newMatrix = tmp;
}

void BlockGpu::prepareData() {
	hipSetDevice(deviceNumber);
	
	dim3 threads ( BLOCK_SIZE );
	dim3 blocksLength  ( (int)ceil((double)length / threads.x) );
	dim3 blocksWidth  ( (int)ceil((double)width / threads.x) );
	
	copyBorderFromMatrix <<< blocksWidth, threads >>> (blockBorderOnDevice, matrix, sendBorderTypeOnDevice, blockBorderMove, TOP, length, width);
	copyBorderFromMatrix <<< blocksLength, threads >>> (blockBorderOnDevice, matrix, sendBorderTypeOnDevice, blockBorderMove, LEFT, length, width);
	copyBorderFromMatrix <<< blocksWidth, threads >>> (blockBorderOnDevice, matrix, sendBorderTypeOnDevice, blockBorderMove, BOTTOM, length, width);
	copyBorderFromMatrix <<< blocksLength, threads >>> (blockBorderOnDevice, matrix, sendBorderTypeOnDevice, blockBorderMove, RIGHT, length, width);
}

double* BlockGpu::getResult() {
	hipSetDevice(deviceNumber);
	
	hipMemcpy( result, matrix, width * length * sizeof(double), hipMemcpyDeviceToHost );
	
	return result;
}

void BlockGpu::print() {
	hipSetDevice(deviceNumber);
	
	double* matrixToPrint = new double [length * width];
	
	int** sendBorderTypeToPrint = new int* [BORDER_COUNT];
	sendBorderTypeToPrint[TOP] = new int [width];
	sendBorderTypeToPrint[LEFT] = new int [length];
	sendBorderTypeToPrint[BOTTOM] = new int [width];
	sendBorderTypeToPrint[RIGHT] = new int [length];
	
	int** receiveBorderTypeToPrint = new int* [BORDER_COUNT];
	receiveBorderTypeToPrint[TOP] = new int [width];
	receiveBorderTypeToPrint[LEFT] = new int [length];
	receiveBorderTypeToPrint[BOTTOM] = new int [width];
	receiveBorderTypeToPrint[RIGHT] = new int [length];
	
	int* blockBorderMoveToPrint = new int [countSendSegmentBorder];
	int* externalBorderMoveToPrint = new int [countReceiveSegmentBorder];
	
	
	hipMemcpy( matrixToPrint, matrix, length * width * sizeof(double), hipMemcpyDeviceToHost );
	
	hipMemcpy( sendBorderTypeToPrint[TOP], sendBorderType[TOP], width * sizeof(int), hipMemcpyDeviceToHost );
	hipMemcpy( sendBorderTypeToPrint[LEFT], sendBorderType[LEFT], length * sizeof(int), hipMemcpyDeviceToHost );
	hipMemcpy( sendBorderTypeToPrint[BOTTOM], sendBorderType[BOTTOM], width * sizeof(int), hipMemcpyDeviceToHost );
	hipMemcpy( sendBorderTypeToPrint[RIGHT], sendBorderType[RIGHT], length * sizeof(int), hipMemcpyDeviceToHost );
	
	hipMemcpy( receiveBorderTypeToPrint[TOP], receiveBorderType[TOP], width * sizeof(int), hipMemcpyDeviceToHost );
	hipMemcpy( receiveBorderTypeToPrint[LEFT], receiveBorderType[LEFT], length * sizeof(int), hipMemcpyDeviceToHost );
	hipMemcpy( receiveBorderTypeToPrint[BOTTOM], receiveBorderType[BOTTOM], width * sizeof(int), hipMemcpyDeviceToHost );
	hipMemcpy( receiveBorderTypeToPrint[RIGHT], receiveBorderType[RIGHT], length * sizeof(int), hipMemcpyDeviceToHost );
	
	hipMemcpy( blockBorderMoveToPrint, blockBorderMove, countSendSegmentBorder * sizeof(int), hipMemcpyDeviceToHost );
	hipMemcpy( externalBorderMoveToPrint, externalBorderMove, countReceiveSegmentBorder * sizeof(int), hipMemcpyDeviceToHost );
	
	
	printf("FROM NODE #%d", nodeNumber);

	printf("\nLength: %d, Width: %d\n", length, width);
	printf("\nlengthMove: %d, widthMove: %d\n", lenghtMove, widthMove);

	printf("\nMatrix:\n");
	for (int i = 0; i < length; ++i)
	{
		for (int j = 0; j < width; ++j)
			printf("%6.1f ", matrixToPrint[i * width + j]);
		printf("\n");
	}
	
	printf("\ntopSendBorderType\n");
	for (int i = 0; i < width; ++i)
		printf("%4d", sendBorderTypeToPrint[TOP][i]);
	printf("\n");

	printf("\nleftSendBorderType\n");
	for (int i = 0; i < length; ++i)
		printf("%4d", sendBorderTypeToPrint[LEFT][i]);
	printf("\n");

	printf("\nbottomSendBorderType\n");
	for (int i = 0; i < width; ++i)
		printf("%4d", sendBorderTypeToPrint[BOTTOM][i]);
	printf("\n");

	printf("\nrightSendBorderType\n");
	for (int i = 0; i < length; ++i)
		printf("%4d", sendBorderTypeToPrint[RIGHT][i]);
	printf("\n\n\n");
	
	
	
	printf("\ntopReceiveBorderType\n");
	for (int i = 0; i < width; ++i)
		printf("%4d", receiveBorderTypeToPrint[TOP][i]);
	printf("\n");

	printf("\nleftReceiveBorderType\n");
	for (int i = 0; i < length; ++i)
		printf("%4d", receiveBorderTypeToPrint[LEFT][i]);
	printf("\n");

	printf("\nbottomReceiveBorderType\n");
	for (int i = 0; i < width; ++i)
		printf("%4d", receiveBorderTypeToPrint[BOTTOM][i]);
	printf("\n");

	printf("\nrightReceiveBorderType\n");
	for (int i = 0; i < length; ++i)
		printf("%4d", receiveBorderTypeToPrint[RIGHT][i]);
	printf("\n");
	
	
	
	for (int i = 0; i < countSendSegmentBorder; ++i)
		printf("\nblockBorder #%d : %d : %d\n", i, blockBorder[i], blockBorderMoveToPrint[i]);

	for (int i = 0; i < countReceiveSegmentBorder; ++i)
		printf("\nexternalBorder #%d : %d : %d\n", i, externalBorder[i], externalBorderMoveToPrint[i]);
	

	printf("\n\n\n");
}

double* BlockGpu::addNewBlockBorder(Block* neighbor, int side, int move, int borderLength) {
	hipSetDevice(deviceNumber);
	
	if( checkValue(side, move + borderLength) ) {
		printf("\nCritical error!\n");
		exit(1);
	}

	dim3 threads ( BLOCK_SIZE );
	dim3 blocks  ( (int)ceil((double)borderLength / threads.x) );
	
	assignIntArray <<< blocks, threads >>> ( sendBorderType[side] + move, countSendSegmentBorder, borderLength );

	countSendSegmentBorder++;

	double* newBlockBorder;

	if( nodeNumber == neighbor->getNodeNumber() ) {
		if( isCPU( neighbor->getBlockType() ) )
			hipHostMalloc ( (void**)&newBlockBorder, borderLength * sizeof(double) );
		
		if( isGPU( neighbor->getBlockType() ) && deviceNumber != neighbor->getDeviceNumber() )
			hipHostMalloc ( (void**)&newBlockBorder, borderLength * sizeof(double) );
	}
	else
		hipMalloc ( (void**)&newBlockBorder, borderLength * sizeof(double) );

	tempBlockBorder.push_back(newBlockBorder);
	tempBlockBorderMove.push_back(move);

	return newBlockBorder;
}

double* BlockGpu::addNewExternalBorder(Block* neighbor, int side, int move, int borderLength, double* border) {
	hipSetDevice(deviceNumber);
	
	if( checkValue(side, move + borderLength) ) {
		printf("\nCritical error!\n");
		exit(1);
	}

	dim3 threads ( BLOCK_SIZE );
	dim3 blocks  ( (int)ceil((double)borderLength / threads.x) );
	
	assignIntArray <<< blocks, threads >>> ( receiveBorderType[side] + move, countReceiveSegmentBorder, borderLength );

	countReceiveSegmentBorder++;

	double* newExternalBorder;

	if( nodeNumber == neighbor->getNodeNumber() )
		newExternalBorder = border;
	else
		hipMalloc ( (void**)&newExternalBorder, borderLength * sizeof(double) );

	tempExternalBorder.push_back(newExternalBorder);
	tempExternalBorderMove.push_back(move);

	return newExternalBorder;
}

void BlockGpu::moveTempBorderVectorToBorderArray() {
	hipSetDevice(deviceNumber);
	
	blockBorder = new double* [countSendSegmentBorder];
	int* tempBlockBorderMoveArray = new int [countSendSegmentBorder];

	externalBorder = new double* [countReceiveSegmentBorder];
	int* tempExternalBorderMoveArray = new int [countReceiveSegmentBorder];

	for (int i = 0; i < countSendSegmentBorder; ++i) {
		blockBorder[i] = tempBlockBorder.at(i);
		tempBlockBorderMoveArray[i] = tempBlockBorderMove.at(i);
	}

	for (int i = 0; i < countReceiveSegmentBorder; ++i) {
		externalBorder[i] = tempExternalBorder.at(i);
		tempExternalBorderMoveArray[i] = tempExternalBorderMove.at(i);
	}

	tempBlockBorder.clear();
	tempBlockBorderMove.clear();
	tempExternalBorder.clear();
	tempExternalBorderMove.clear();
	
	hipMalloc ( (void**)&blockBorderOnDevice, countSendSegmentBorder * sizeof(double*) );
	hipMemcpy( blockBorderOnDevice, blockBorder, countSendSegmentBorder * sizeof(double*), hipMemcpyHostToDevice );
	
	hipMalloc ( (void**)&externalBorderOnDevice, countReceiveSegmentBorder * sizeof(double*) );
	hipMemcpy( externalBorderOnDevice, externalBorder, countReceiveSegmentBorder * sizeof(double*), hipMemcpyHostToDevice );
	
	hipMalloc ( (void**)&blockBorderMove, countSendSegmentBorder * sizeof(int) );
	hipMemcpy( blockBorderMove, tempBlockBorderMoveArray, countSendSegmentBorder * sizeof(int), hipMemcpyHostToDevice );
	
	hipMalloc ( (void**)&externalBorderMove, countReceiveSegmentBorder * sizeof(int) );	
	hipMemcpy( externalBorderMove, tempExternalBorderMoveArray, countReceiveSegmentBorder * sizeof(int), hipMemcpyHostToDevice );
	
	delete tempBlockBorderMoveArray;
	delete tempExternalBorderMoveArray;
}