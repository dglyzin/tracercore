/*
 * BlockCpu.cpp
 *
 *  Created on: 20 янв. 2015 г.
 *      Author: frolov
 */

#include "blockcpu.h"

using namespace std;

BlockCpu::BlockCpu(int _length, int _width, int _lengthMove, int _widthMove, int _nodeNumber, int _deviceNumber) : Block(  _length, _width, _lengthMove, _widthMove, _nodeNumber, _deviceNumber  ) {

	matrix = new double [length * width];
	newMatrix = new double [length * width];

	for (int i = 0; i < length; ++i)
		for (int j = 0; j < width; ++j)
			matrix[i * width + j] = newMatrix[i * width + j] = 0;

	/*
	 * Типы границ блока. Выделение памяти.
	 * По умолчанию границы задаются функциями, то есть нет границ между блоками.
	 */
	sendBorderType = new int* [BORDER_COUNT];

	sendBorderType[TOP] = new int[width];
	for(int i = 0; i < width; i++)
		sendBorderType[TOP][i] = BY_FUNCTION;

	sendBorderType[LEFT] = new int[length];
	for (int i = 0; i < length; ++i)
		sendBorderType[LEFT][i] = BY_FUNCTION;

	sendBorderType[BOTTOM] = new int[width];
	for(int i = 0; i < width; i++)
		sendBorderType[BOTTOM][i] = BY_FUNCTION;

	sendBorderType[RIGHT] = new int[length];
	for (int i = 0; i < length; ++i)
		sendBorderType[RIGHT][i] = BY_FUNCTION;


	receiveBorderType = new int* [BORDER_COUNT];

	receiveBorderType[TOP] = new int[width];
	for(int i = 0; i < width; i++)
		receiveBorderType[TOP][i] = BY_FUNCTION;

	receiveBorderType[LEFT] = new int[length];
	for (int i = 0; i < length; ++i)
		receiveBorderType[LEFT][i] = BY_FUNCTION;

	receiveBorderType[BOTTOM] = new int[width];
	for(int i = 0; i < width; i++)
		receiveBorderType[BOTTOM][i] = BY_FUNCTION;

	receiveBorderType[RIGHT] = new int[length];
	for (int i = 0; i < length; ++i)
		receiveBorderType[RIGHT][i] = BY_FUNCTION;
	
	
	result = new double [length * width];
}

BlockCpu::~BlockCpu() {
	if(matrix != NULL)
		delete matrix;
	
	if(newMatrix != NULL)
		delete newMatrix;
	
	if(sendBorderType != NULL) {
		if(sendBorderType[TOP] != NULL)
			delete sendBorderType[TOP];
		
		if(sendBorderType[LEFT] != NULL)
			delete sendBorderType[LEFT];
		
		if(sendBorderType[BOTTOM] != NULL)
			delete sendBorderType[BOTTOM];
		
		if(sendBorderType[RIGHT] != NULL)
			delete sendBorderType[RIGHT];
		
		delete sendBorderType;		
	}
	
	if(receiveBorderType != NULL) {
		if(receiveBorderType[TOP] != NULL)
			delete receiveBorderType[TOP];
		
		if(receiveBorderType[LEFT] != NULL)
			delete receiveBorderType[LEFT];
		
		if(receiveBorderType[BOTTOM] != NULL)
			delete receiveBorderType[BOTTOM];
		
		if(receiveBorderType[RIGHT] != NULL)
			delete receiveBorderType[RIGHT];
		
		delete receiveBorderType;		
	}
	
	
	if(blockBorder != NULL) {
		for(int i = 0; i < countSendSegmentBorder; i++ )
			freeMemory(blockBorderMemoryAllocType[i], blockBorder[i]);
		
		delete blockBorder;
		delete blockBorderMemoryAllocType;
	}
	
	if(blockBorderMove != NULL)
		delete blockBorderMove;
	
	
	if(externalBorder != NULL) {
		for(int i = 0; i < countReceiveSegmentBorder; i++ )
			freeMemory(externalBorderMemoryAllocType[i], externalBorder[i]);
		
		delete externalBorder;
		delete externalBorderMemoryAllocType;
	}
	
	if(externalBorderMove != NULL)
		delete externalBorderMove;
	
	if(result != NULL)
		delete result;
}

void BlockCpu::computeOneStep(double dX2, double dY2, double dT) {
	/*
	 * Теплопроводность
	 */

	/*
	 * Параллельное вычисление на максимально возможном количестве потоков.
	 * Максимально возможное количесвто потоков получается из-за самой библиотеки omp
	 * Если явно не указывать, какое именно количесвто нитей необходимо создать, то будет создано макстимально возможное на данный момент.
	 */
# pragma omp parallel
	{
		/*
		 * Для решения задачи теплопроводности нам необходимо знать несколько значений.
		 * Среди них
		 * значение в ячейке выше
		 * значение в ячейке слева
		 * значение в ячейке снизу
		 * значение в ячейке справа
		 * текущее значение в данной ячейке
		 *
		 * остально данные передаются в функцию в качестве параметров.
		 */
	double top, left, bottom, right, cur;

# pragma omp for
	/*
	 * Проходим по всем ячейкам матрицы.
	 * Для каждой из них будет выполнен перерасчет.
	 */
	for (int i = 0; i < length; ++i)
		for (int j = 0; j < width; ++j) {
			/*
			 * Если находимся на верхней границе блока.
			 * В таком случае необходимо проверить тип границы и в зависимости от ответа принать решение.
			 *
			 * Стоит отличать границу реальную от границы с блоком.
			 * Если граница реальна, то точка на границе может не иметь значения выше / значения ниже и так далее, так как это реально границе ВСЕЙ ОБЛАСТИ.
			 * Если эта граница с другим блоком, то значение выше / ниже сущесвтуют, так как это не граница области.
			 * Значит их нужно получить и использовать при ирасчете нового значения.
			 */
			
			if( i == 0 )
				/*
				 * На данный момент есть только 2 типа границы. Функция и другой блок.
				 * Поэтому использование else корректно.
				 *
				 * Если граница задана функцией, то это значит,
				 * что наданном этапе в массиве externalBorder уже должны лежать свежие данные от функции.
				 * В таком случае просто копируем данные из массива в матрицу. Для этой ячейки расчет окончен.
				 *
				 * Если это граница с другим блоком, то в top (значение в ячейке выше данной) записываем информацию с гранцы.
				 * Но продолжаем расчет.
				 */
				if( receiveBorderType[TOP][j] == BY_FUNCTION ) {
					newMatrix[i * width + j] = 100;
					continue;
				}
				else
					top = externalBorder[	receiveBorderType[TOP][j]	][j - externalBorderMove[	receiveBorderType[TOP][j]	]];
			else
				/*
				 * Если находимся не на верхней границе блока, то есть возможность просто получить значение в ячейке выше данной.
				 */
				top = matrix[(i - 1) * width + j];


			/*
			 * Аналогично предыдущему случаю.
			 * Только здесь проверка на левую границу блока.
			 *
			 * Рассуждения полностью совпадают со случаем верхней границы.
			 */
			if( j == 0 )
				if( receiveBorderType[LEFT][i] == BY_FUNCTION ) {
					newMatrix[i * width + j] = 10;
					continue;
				}
				else
					left = externalBorder[	receiveBorderType[LEFT][i]	][i - externalBorderMove[	receiveBorderType[LEFT][i]		]];
			else
				left = matrix[i * width + (j - 1)];


			/*
			 * Аналогично первому случаю.
			 * Граница нижняя.
			 */
			if( i == length - 1 )
				if( receiveBorderType[BOTTOM][j] == BY_FUNCTION ) {
					newMatrix[i * width + j] = 10;
					continue;
				}
				else
					bottom = externalBorder[	receiveBorderType[BOTTOM][j]	][j - externalBorderMove[	receiveBorderType[BOTTOM][j]	]];
			else
				bottom = matrix[(i + 1) * width + j];


			/*
			 * Аналогично первому случаю.
			 * Граница правая.
			 */
			if( j == width - 1 )
				if( receiveBorderType[RIGHT][i] == BY_FUNCTION ) {
					newMatrix[i * width + j] = 10;
					continue;
				}
				else
					right = externalBorder[	receiveBorderType[RIGHT][i]	][i - externalBorderMove[	receiveBorderType[RIGHT][i]	]];
			else
				right = matrix[i * width + (j + 1)];


			/*
			 * Текущее значение всегда (если вообще дошли до этого места) можно просто получить из матрицы.
			 */
			cur = matrix[i * width + j];

			/*
			 * Формула расчета для конкретной точки.
			 */
			newMatrix[i * width + j] = cur + dT * ( ( left - 2*cur + right )/dX2 + ( top - 2*cur + bottom )/dY2  );
		}
	}
/*
 * Указатель на старую матрицу запоминается
 * Новая матрица становится текущей
 * Память, занимаемая старой матрицей освобождается.
 */
	double* tmp = matrix;

	matrix = newMatrix;

	newMatrix = tmp;
}

void BlockCpu::computeOneStepBorder(double dX2, double dY2, double dT) {
	/*
	 * Теплопроводность
	 */

	/*
	 * Параллельное вычисление на максимально возможном количестве потоков.
	 * Максимально возможное количесвто потоков получается из-за самой библиотеки omp
	 * Если явно не указывать, какое именно количесвто нитей необходимо создать, то будет создано макстимально возможное на данный момент.
	 */
# pragma omp parallel
	{
		/*
		 * Для решения задачи теплопроводности нам необходимо знать несколько значений.
		 * Среди них
		 * значение в ячейке выше
		 * значение в ячейке слева
		 * значение в ячейке снизу
		 * значение в ячейке справа
		 * текущее значение в данной ячейке
		 *
		 * остально данные передаются в функцию в качестве параметров.
		 */
	double top, left, bottom, right, cur;

# pragma omp for
	/*
	 * Проходим по всем ячейкам матрицы.
	 * Для каждой из них будет выполнен перерасчет.
	 */
	for (int i = 0; i < length; ++i)
		for (int j = 0; j < width; ++j) {
			/*
			 * Если находимся на верхней границе блока.
			 * В таком случае необходимо проверить тип границы и в зависимости от ответа принать решение.
			 *
			 * Стоит отличать границу реальную от границы с блоком.
			 * Если граница реальна, то точка на границе может не иметь значения выше / значения ниже и так далее, так как это реально границе ВСЕЙ ОБЛАСТИ.
			 * Если эта граница с другим блоком, то значение выше / ниже сущесвтуют, так как это не граница области.
			 * Значит их нужно получить и использовать при ирасчете нового значения.
			 */
			
			if( i == 0 )
				/*
				 * На данный момент есть только 2 типа границы. Функция и другой блок.
				 * Поэтому использование else корректно.
				 *
				 * Если граница задана функцией, то это значит,
				 * что наданном этапе в массиве externalBorder уже должны лежать свежие данные от функции.
				 * В таком случае просто копируем данные из массива в матрицу. Для этой ячейки расчет окончен.
				 *
				 * Если это граница с другим блоком, то в top (значение в ячейке выше данной) записываем информацию с гранцы.
				 * Но продолжаем расчет.
				 */
				if( receiveBorderType[TOP][j] == BY_FUNCTION ) {
					newMatrix[i * width + j] = 100;
					continue;
				}
				else
					top = externalBorder[	receiveBorderType[TOP][j]	][j - externalBorderMove[	receiveBorderType[TOP][j]	]];


			/*
			 * Аналогично предыдущему случаю.
			 * Только здесь проверка на левую границу блока.
			 *
			 * Рассуждения полностью совпадают со случаем верхней границы.
			 */
			if( j == 0 )
				if( receiveBorderType[LEFT][i] == BY_FUNCTION ) {
					newMatrix[i * width + j] = 10;
					continue;
				}
				else
					left = externalBorder[	receiveBorderType[LEFT][i]	][i - externalBorderMove[	receiveBorderType[LEFT][i]		]];


			/*
			 * Аналогично первому случаю.
			 * Граница нижняя.
			 */
			if( i == length - 1 )
				if( receiveBorderType[BOTTOM][j] == BY_FUNCTION ) {
					newMatrix[i * width + j] = 10;
					continue;
				}
				else
					bottom = externalBorder[	receiveBorderType[BOTTOM][j]	][j - externalBorderMove[	receiveBorderType[BOTTOM][j]	]];


			/*
			 * Аналогично первому случаю.
			 * Граница правая.
			 */
			if( j == width - 1 )
				if( receiveBorderType[RIGHT][i] == BY_FUNCTION ) {
					newMatrix[i * width + j] = 10;
					continue;
				}
				else
					right = externalBorder[	receiveBorderType[RIGHT][i]	][i - externalBorderMove[	receiveBorderType[RIGHT][i]	]];

			cur = matrix[i * width + j];

			newMatrix[i * width + j] = cur + dT * ( ( left - 2*cur + right )/dX2 + ( top - 2*cur + bottom )/dY2  );
		}
	}
}

void BlockCpu::computeOneStepCenter(double dX2, double dY2, double dT) {
	/*
	 * Теплопроводность
	 */

	/*
	 * Параллельное вычисление на максимально возможном количестве потоков.
	 * Максимально возможное количесвто потоков получается из-за самой библиотеки omp
	 * Если явно не указывать, какое именно количесвто нитей необходимо создать, то будет создано макстимально возможное на данный момент.
	 */
# pragma omp parallel
	{
		/*
		 * Для решения задачи теплопроводности нам необходимо знать несколько значений.
		 * Среди них
		 * значение в ячейке выше
		 * значение в ячейке слева
		 * значение в ячейке снизу
		 * значение в ячейке справа
		 * текущее значение в данной ячейке
		 *
		 * остально данные передаются в функцию в качестве параметров.
		 */
	double top, left, bottom, right, cur;

# pragma omp for
	/*
	 * Проходим по всем ячейкам матрицы.
	 * Для каждой из них будет выполнен перерасчет.
	 */
	for (int i = 1; i < length - 1; ++i)
		for (int j = 1; j < width - 1; ++j) {
			top = matrix[(i - 1) * width + j];
			left = matrix[i * width + (j - 1)];
			bottom = matrix[(i + 1) * width + j];
			right = matrix[i * width + (j + 1)];

			cur = matrix[i * width + j];

			newMatrix[i * width + j] = cur + dT * ( ( left - 2*cur + right )/dX2 + ( top - 2*cur + bottom )/dY2  );
		}
	}
}

void BlockCpu::prepareData() {
	/*
	 * Копирование данных из матрицы в массивы.
	 * В дальнейшем эти массивы будет пеесылаться другим блокам.
	 */
	for (int i = 0; i < width; ++i)
		if( sendBorderType[TOP][i] != BY_FUNCTION )
			blockBorder[	sendBorderType[TOP][i]	][i - blockBorderMove[	sendBorderType[TOP][i]	]] = matrix[0 * width + i];

	for (int i = 0; i < length; ++i)
		if( sendBorderType[LEFT][i] != BY_FUNCTION )
			blockBorder[	sendBorderType[LEFT][i]	][i - blockBorderMove[	sendBorderType[LEFT][i]	]] = matrix[i * width + 0];

	for (int i = 0; i < width; ++i)
		if( sendBorderType[BOTTOM][i] != BY_FUNCTION )
			blockBorder[	sendBorderType[BOTTOM][i]	][i - blockBorderMove[	sendBorderType[BOTTOM][i]	]] = matrix[(length - 1) * width + i];

	for (int i = 0; i < length; ++i)
		if( sendBorderType[RIGHT][i] != BY_FUNCTION )
			blockBorder[	sendBorderType[RIGHT][i]	][i - blockBorderMove[	sendBorderType[RIGHT][i]	]] = matrix[i * width + (width - 1)];
}

double* BlockCpu::getResult() {
	for(int i = 0; i < length * width; i++)
		result[i] = matrix[i];
	
	return result;
}

void BlockCpu::print() {
	cout << "########################################################################################################################################################################################################" << endl;
	
	cout << endl;
	cout << "BlockCpu from node #" << nodeNumber << endl;
	cout << "Length:      " << length << endl;
	cout << "Width :      " << width << endl;
	cout << endl;
	cout << "Length move: " << lengthMove << endl;
	cout << "Width move:  " << widthMove << endl;
	
	cout << endl;
	cout << "Block matrix:" << endl;
	cout.setf(ios::fixed);
	for(int i = 0; i < length; i++) {
		for( int j = 0; j < width; j++ ) {
			cout.width(7);
			cout.precision(1);
			cout << matrix[i * width + j];
		}
		cout << endl;
	}
	
	cout << endl;
	cout << "TopSendBorderType" << endl;
	for( int i =0; i < width; i++ ) {
		cout.width(4);
		cout << sendBorderType[TOP][i] << " ";
	}
	cout << endl;

	cout << endl;
	cout << "LeftSendBorderType" << endl;
	for( int i =0; i < length; i++ ) {
		cout.width(4);
		cout << sendBorderType[LEFT][i] << " ";
	}
	cout << endl;

	cout << endl;
	cout << "BottomSendBorderType" << endl;
	for( int i =0; i < width; i++ ) {
		cout.width(4);
		cout << sendBorderType[BOTTOM][i] << " ";
	}
	cout << endl;

	cout << endl;
	cout << "RightSendBorderType" << endl;
	for( int i =0; i < length; i++ ) {
		cout.width(4);
		cout << sendBorderType[RIGHT][i] << " ";
	}
	cout << endl;

	
	cout << endl << endl;

	
	cout << endl;
	cout << "TopRecieveBorderType" << endl;
	for( int i =0; i < width; i++ ) {
		cout.width(4);
		cout << receiveBorderType[TOP][i] << " ";
	}
	cout << endl;

	cout << endl;
	cout << "LeftRecieveBorderType" << endl;
	for( int i =0; i < length; i++ ) {
		cout.width(4);
		cout << receiveBorderType[LEFT][i] << " ";
	}
	cout << endl;

	cout << endl;
	cout << "BottomRecieveBorderType" << endl;
	for( int i =0; i < width; i++ ) {
		cout.width(4);
		cout << receiveBorderType[BOTTOM][i] << " ";
	}
	cout << endl;

	cout << endl;
	cout << "RightRecieveBorderType" << endl;
	for( int i =0; i < length; i++ ) {
		cout.width(4);
		cout << receiveBorderType[RIGHT][i] << " ";
	}
	cout << endl;

	
	cout << endl << endl;

	
	cout << endl;
	for (int i = 0; i < countSendSegmentBorder; ++i) {
		cout << "BlockBorder #" << i << endl;
		cout << "	Memory address: " << blockBorder[i] << endl;
		cout << "	Border move:    " << blockBorderMove[i] << endl;
		cout << endl;
	}
	
	
	cout << endl;
	
		
	cout << endl;
	for (int i = 0; i < countReceiveSegmentBorder; ++i) {
		cout << "ExternalBorder #" << i << endl;
		cout << "	Memory address: " << externalBorder[i] << endl;
		cout << "	Border move:    " << externalBorderMove[i] << endl;
		cout << endl;
	}

	cout << "########################################################################################################################################################################################################" << endl;
	cout << endl << endl;
}

double* BlockCpu::addNewBlockBorder(Block* neighbor, int side, int move, int borderLength) {
	if( checkValue(side, move + borderLength) ) {
		printf("\nCritical error!\n");
		exit(1);
	}

	for (int i = 0; i < borderLength; ++i)
		sendBorderType[side][i + move] = countSendSegmentBorder;

	countSendSegmentBorder++;

	double* newBlockBorder;

	if( ( nodeNumber == neighbor->getNodeNumber() ) && isGPU( neighbor->getBlockType() ) ) {
		hipHostMalloc ( (void**)&newBlockBorder, borderLength * sizeof(double) );
		tempBlockBorderMemoryAllocType.push_back(CUDA_MALLOC_HOST);
	}
	else {
		newBlockBorder = new double [borderLength];
		tempBlockBorderMemoryAllocType.push_back(NEW);
	}

	tempBlockBorder.push_back(newBlockBorder);
	tempBlockBorderMove.push_back(move);

	return newBlockBorder;
}

double* BlockCpu::addNewExternalBorder(Block* neighbor, int side, int move, int borderLength, double* border) {
	if( checkValue(side, move + borderLength) ) {
		printf("\nCritical error!\n");
		exit(1);
	}

	for (int i = 0; i < borderLength; ++i)
		receiveBorderType[side][i + move] = countReceiveSegmentBorder;

	countReceiveSegmentBorder++;

	double* newExternalBorder;

	if( nodeNumber == neighbor->getNodeNumber() ) {
		newExternalBorder = border;
		tempExternalBorderMemoryAllocType.push_back(NOT_ALLOC);
	}
	else {
		newExternalBorder = new double [borderLength];
		tempExternalBorderMemoryAllocType.push_back(NEW);
	}

	tempExternalBorder.push_back(newExternalBorder);
	tempExternalBorderMove.push_back(move);

	return newExternalBorder;
}

void BlockCpu::moveTempBorderVectorToBorderArray() {
	blockBorder = new double* [countSendSegmentBorder];
	blockBorderMove = new int [countSendSegmentBorder];
	blockBorderMemoryAllocType = new int [countSendSegmentBorder];

	externalBorder = new double* [countReceiveSegmentBorder];
	externalBorderMove = new int [countReceiveSegmentBorder];
	externalBorderMemoryAllocType = new int [countReceiveSegmentBorder];	
	

	for (int i = 0; i < countSendSegmentBorder; ++i) {
		blockBorder[i] = tempBlockBorder.at(i);
		blockBorderMove[i] = tempBlockBorderMove.at(i);
		blockBorderMemoryAllocType[i] = tempBlockBorderMemoryAllocType.at(i);
	}

	for (int i = 0; i < countReceiveSegmentBorder; ++i) {
		externalBorder[i] = tempExternalBorder.at(i);
		externalBorderMove[i] = tempExternalBorderMove.at(i);
		externalBorderMemoryAllocType[i] = tempExternalBorderMemoryAllocType.at(i);
	}

	tempBlockBorder.clear();
	tempBlockBorderMove.clear();
	tempExternalBorder.clear();
	tempExternalBorderMove.clear();
	
	tempBlockBorderMemoryAllocType.clear();
	tempExternalBorderMemoryAllocType.clear();
}

void BlockCpu::loadData(double* data) {
	for(int i = 0; i < length * width; i++)
		matrix[i] = data[i];
}
